#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <time.h>
//#include "MDR.h"
//#include "MDR_kernel.cu"
//#include "MDR.cu"

#if _WIN32
    //Windows threads.
    #include <windows.h>

    typedef HANDLE CUTThread;
    typedef unsigned (WINAPI *CUT_THREADROUTINE)(void *);

    #define CUT_THREADPROC unsigned WINAPI
    #define  CUT_THREADEND return 0

#else
    //POSIX threads.
    #include <pthread.h>

    typedef pthread_t CUTThread;
    typedef void *(*CUT_THREADROUTINE)(void *);

    #define CUT_THREADPROC void
    #define  CUT_THREADEND
#endif

//Create thread.
CUTThread start_thread( CUT_THREADROUTINE, void *data );

//Wait for thread to finish.
void end_thread( CUTThread thread );

//Destroy thread.
void destroy_thread( CUTThread thread );

//Wait for multiple threads.
void wait_for_threads( const CUTThread *threads, int num );

#if _WIN32
    //Create thread
    CUTThread start_thread(CUT_THREADROUTINE func, void *data){
        return CreateThread(NULL, 0, (LPTHREAD_START_ROUTINE)func, data, 0, NULL);
    }

    //Wait for thread to finish
    void end_thread(CUTThread thread){
        WaitForSingleObject(thread, INFINITE);
        CloseHandle(thread);
    }

    //Destroy thread
    void destroy_thread( CUTThread thread ){
        TerminateThread(thread, 0);
        CloseHandle(thread);
    }

    //Wait for multiple threads
    void wait_for_threads(const CUTThread * threads, int num){
        WaitForMultipleObjects(num, threads, true, INFINITE);

        for(int i = 0; i < num; i++)
            CloseHandle(threads[i]);
    }

#else
    //Create thread
    CUTThread start_thread(CUT_THREADROUTINE func, void * data){
        pthread_t thread;
        pthread_create(&thread, NULL, func, data);
        return thread;
    }

    //Wait for thread to finish
    void end_thread(CUTThread thread){
        pthread_join(thread, NULL);
    }

    //Destroy thread
    void destroy_thread( CUTThread thread ){
        pthread_cancel(thread);
    }

    //Wait for multiple threads
    void wait_for_threads(const CUTThread * threads, int num){
        for(int i = 0; i < num; i++)
            end_thread( threads[i] );
    }

#endif




#define imin(a,b) (a<b?a:b)
#define imax(a,b) (a>b?a:b)
#define func(a)

char* phenoFile;
char* genoFile;
char* outputFile;
char* combFile;
int basic_model;

float THR = THR ; //leave this space
#define NSNPS NSNPS
#define NUMCOMBS NUMCOMBS
#define CUT CUT
#define NIND NIND
#define BSx BSx
#define ORDER ORDER
#define CV CV
#define TABLE_SIZE TABLE_SIZE
#define NUMDEVICES NUMDEVICES
#define ONEORTWO ONEORTWO //two if CV > 1
#define GSx ((NUMCOMBS/NUMDEVICES+BSx-1) / BSx )
#define MEASURE 'MEASURE'

#define mat_SNP_size NIND * NSNPS * sizeof(int)
#define v_pheno_size NIND * sizeof(int)
#define output_size NUMCOMBS *  ONEORTWO * CV * sizeof(float) //oneotwo: 2 is one for train and one for test
#define fp_size NUMCOMBS * sizeof(int)
#define tp_size NUMCOMBS * sizeof(int)
#define combinations_size NUMCOMBS * ORDER * sizeof(int)
#define indices_size NIND * sizeof(int)


#define TESTCOMB -112
#define TESTSNP0 -87509
#define TESTSNP1 -370675

struct controlscases {
int controls;
int cases;
};


 struct str
{
   float value;
   int index;
 };

struct DataStruct {
    int		deviceID;
    int		deviceCount;
    int*	mat_SNP;
    int*	combinations;
    int* 	v_pheno;
    int* 	cv_indices;
    float*	output;
    int*	tp;
    int*	fp;
    float	start_clock;
};


__device__ float compute_measure(int cases_high, int controls_high, int controls_low, int cases_low, char m){
	float train_measure;
	//int positives = cases_high + cases_low
	//int negatives = controls_low + controls_high
	if (m - '0' == 50){ //BALANCED ACCURACY
		train_measure = (cases_high/float(cases_high + cases_low) + controls_low/float(controls_low + controls_high))/2;
	}
	else if (m - '0' == 49){ //ACCURACY
		train_measure = float(cases_high + controls_low)/float(cases_high + controls_low + controls_high + cases_low);
		
	}
	else if (m - '0' == 55){ //TODO
		train_measure = float(cases_high + controls_low)/float(cases_high + controls_low + controls_high + cases_low);
		
	}
	return train_measure;
}

__device__ int dev_pow(int b,int e) {
	int o = 1;
	if (e == 0)
		return 1;
	for (int i=0; i<e; i++){
		o *= b;
	}
	return o;
}

__device__ int count_digits(int i) {
	if (i < 10)
		return 1;
	if (i < 100)
		return 2;
	if (i < 1000)
		return 3;
	if (i < 10000)
		return 4;
	if (i < 100000)
		return 5;
	return 6;
}

//base decimal to base 3
__device__ void int_to_index(int n, int order, int* v){
	int r;
	for(int i=0; i<order; i++){
		float num = (float)n;
		int den = dev_pow(3,order-i-1);
		r = (int)(num/den);
		n -= r * dev_pow(3,order-i-1);
		v[i] = r ;
	}
	return;
}

//base 3 to base decimal
__device__ int index_to_int(int* v_indices, int order){
	int o = 0;
	for(int i=0; i<order; i++){
		o += v_indices[i] * dev_pow(3, order-i-1);
	}
	return o;
}


//#include "MDR.h"
//#include "MDR.cu"

__constant__ int dev_v_pheno[NIND];
__constant__ int dev_cv_indices[NIND];
	

__global__ void MDR( int* dev_SNP_values, float* dev_output, int* dev_tp, int* dev_fp, int* dev_combinations, float THR, int deviceID, int deviceCount) {
    
    	
	//printf(" %d + %d * %d :", threadIdx.x, blockIdx.x, blockDim.x);
	//__shared__ float cache[BS][threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	
	
	
	//printf(" %d ", tid);
	int d;
	if (deviceID + 1 < deviceCount)
		d = (NUMCOMBS/deviceCount);
	else
		d = NUMCOMBS - 1 - (((0 + ((NUMCOMBS *  ONEORTWO * CV)/(deviceCount) * deviceID ))) - 1); //how many total - how many done
	
	if (tid >= d)
		return;

	//printf(" %d ", tid);
	//int* thread_combination = (int*)malloc(ORDER * sizeof(int));
	int thread_combination[ORDER]; //a combination (thread level)
	//retrieve the combination indices
	for (int i=0; i< ORDER; i++) {
		*(&thread_combination[0] + i) = *(dev_combinations + tid * ORDER + i);
	}
	
	
	//printf("thread with tid %d is assigned combination: <%d, %d>\n", tid, thread_combination[0], thread_combination[1]); 
	
	//retrieve the genotype of each snp in the combination, from SNPvalues, for ALL individuals
	//int* thread_geno = (int*)malloc(NIND * ORDER * sizeof(int));
	/*
	int thread_geno[ORDER * NIND];
	for (int i=0; i< NIND; i++) {
		for (int j=0; j< ORDER; j++) {
			*(&thread_geno[0] + j * NIND + i  ) = *(dev_SNP_values + NIND * *(&thread_combination[0] + j) + i);
		}
	}
	*/
	
	/*
	if (*(&thread_combination[0] + 0) == TESTSNP0 && *(&thread_combination[0] + 1) == TESTSNP1){
		printf("START of SNP_values\n");
		for (int j=0; j < 4050; j++) {
			printf("%d ", *(dev_SNP_values + NIND * 0 + j));
			if (j == 1999 || j == 4000)
				printf("\n");
			}
		printf("\nend START\n\n");
		}
		
	*/
	
	/*
	if (*(&thread_combination[0] + 0) == TESTSNP0 && *(&thread_combination[0] + 1) == TESTSNP1){
		printf("GENOs for %d and %d\n", TESTSNP0, TESTSNP1);
		for (int j=0; j < ORDER; j++) {
			printf("\n");
			for (int i=0; i < NIND; i++) {
				if (i< 10 || i > NIND-10)
					printf("%d ", *(dev_SNP_values + NIND * *(&thread_combination[0] + j) + i));
			}
		}
		printf("\nend GENOs");
	}
	*/
	
	
	struct controlscases thread_table[TABLE_SIZE];

	//replace this initialization?
	for (int i=0; i< TABLE_SIZE; i++) {
		(*(&thread_table[0] + i )).controls = 0;
		(*(&thread_table[0] + i )).cases = 0;
		}
		
	int geno[ORDER]; //support variable for counting, stores a geno combination
	int index_in_table = 0; //from a geno, to the index in the table
	int indiv;
	int cases_high;
	int controls_high;
	int cases_low;
	int controls_low;
	// ba = ((tp/p) + (tn/n))/2 = (sensitivity + specificity) /2
	float train_measure;
	float test_measure;
	int v[ORDER];
	int ph;
	//CV loop
	for (int cv=0; cv<CV; cv++){

		
		
		//*****************
		//TRAINING
		//*****************
		//populate the 3^ORDER-tot-entries table
		for (int n=0; n< NIND; n++) {
			if (CV > 1){
				if ((n >= int((cv/float(CV))*NIND)) && (n <= int(((cv+1)/float(CV))*NIND)) )//reserved for testing
				 		continue;
				 }
			 indiv = *(&dev_cv_indices[0] + n);
			 for (int i=0; i< ORDER; i++) 
			 	geno[i] = *(dev_SNP_values + NIND * *(&thread_combination[0] + i) + indiv); //i-th snp geno
			 index_in_table = index_to_int(geno, ORDER);
			 
			 if (int(*(dev_v_pheno + indiv))) { //get the pheno
			 	
			 	
			 	(*(&thread_table[0] + index_in_table )).cases += 1;
			 }
			 else{
			 	//if (tid == TESTCOMB)
			 	//	printf("?????? geno %d and healthy ph: %d\n",index_in_table, indiv);
			 	(*(&thread_table[0] + index_in_table )).controls += 1;
			 }
		}

		//only a print
		if (tid == TESTCOMB || (*(&thread_combination[0] + 0) == TESTSNP0 && *(&thread_combination[0] + 1) == TESTSNP1)){
			printf("***************\ngpu%d-tid%d\ncomb. ", deviceID, tid);
			for (int q=0; q< ORDER; q++)
				printf("%d ", *(&thread_combination[0] + q));
			printf("\n\n");
			for (int i=0; i< TABLE_SIZE; i++) {
				printf("thread_table[%d].controls, cases: %d %d ",i,(*(&thread_table[0] + i )).controls, (*(&thread_table[0] + i )).cases);
				if ( (((*(&thread_table[0] + i )).cases) / float((*(&thread_table[0] + i )).controls + 0.01) >= THR ) ){
					int_to_index(i, ORDER, v);
					printf(" geno ");
					for (int l=0; l< ORDER; l++)
						printf("%d ", v[l] );
					printf("is HIGH\n");
				}
				else
					printf("\n");
			
			}
			printf("\n");
		}
	
		//moving two a two-dim variable
		cases_high = 0;
		controls_high = 0;
		cases_low = 0;
		controls_low = 0;
		//int c = 0;
		for (int i=0; i< TABLE_SIZE; i++) {
			int_to_index(i, ORDER, v);
				
			if ( (((*(&thread_table[0] + i )).cases) / float((*(&thread_table[0] + i )).controls + 0.01) >= THR )) {
				cases_high += (*(&thread_table[0] + i )).cases;
				controls_high += (*(&thread_table[0] + i )).controls;

			}
			else{
				//here in LOW also the case 0 controls 0 cases
				cases_low += (*(&thread_table[0] + i )).cases;
				controls_low += (*(&thread_table[0] + i )).controls;
				if (tid == TESTCOMB){
					printf("tid %d (comb. ", tid) ;
					for (int q=0; q< ORDER; q++)
						printf("%d ", *(&thread_combination[0] + q));
					printf("), geno ");
					for (int l=0; l< ORDER; l++)
						printf("%d ", v[l] );
					printf("is LOW\n");
				}

			}
		}
		/*
		if (CV > 1)
			*(&high_genos[0] + c*ORDER + 0) = 9; //end sequence, since high_genos only reports the high ones
		*/
		//printf("******************\n");
		
		train_measure = compute_measure(cases_high, controls_high, controls_low, cases_low, MEASURE);
		
		
		
		//only a print
		if (tid == TESTCOMB || (*(&thread_combination[0] + 0) == TESTSNP0 && *(&thread_combination[0] + 1) == TESTSNP1)){
		if (MEASURE - '0' == 50){
		printf("(tid %d) TRAIN BA %1.5f = (%d/(%d+%d) + %d/(%d+%d))/2\n***************\n", 
				 tid, train_measure, cases_high, cases_high, controls_high, controls_low, controls_low, cases_low);
		}
		else if (MEASURE - '0' == 49){
		printf("(tid %d) TRAIN AC	 %1.5f = (%d+%d)/(%d+%d+%d+%d)\n***************\n", 
			 tid, train_measure, cases_high, controls_low, cases_high, controls_low, controls_high, cases_low);
		}
		else if (MEASURE - '0' == 55){
		printf("(tid %d) TRAIN AC	 %1.5f = (%d+%d)/(%d+%d+%d+%d)\n***************\n", 
			 tid, train_measure, cases_high, controls_low, cases_high, controls_low, controls_high, cases_low);
		
		}
		}
		
		
		
		
		

		//write result to global memory
		if (CV == 1){
			*(dev_output + NUMCOMBS * 0 + 1 * tid + 0) = train_measure;
			*(dev_tp + NUMCOMBS * 0 + 1 * tid + 0) = cases_high;
			*(dev_fp + NUMCOMBS * 0 + 1 * tid + 0) = controls_high;
			}
		else
			*(dev_output + NUMCOMBS * cv + 2 * tid + 0) = train_measure;
			//TODO
		
		
		if (CV > 1)
		{ //TODO
			//*****************
			//TESTING
			//*****************
		
			if (tid == TESTCOMB) printf("CV-TEST %d/%d:\n", cv+1, CV);
		
			cases_high = 0;
			controls_high = 0;
			cases_low = 0;
			controls_low = 0;
			for (int n=0; n< NIND; n++) {
				 if ((n < int((cv/float(CV))*NIND)) || (n > int(((cv+1)/float(CV))*NIND)) )//reserved for training
				 	continue;
				 
				 indiv = *(&dev_cv_indices[0] + n);
				 for (int i=0; i< ORDER; i++) 
				 	geno[i] = *(dev_SNP_values + NIND * *(&thread_combination[0] + i) + indiv); //i-th snp geno
				 ph = int(*(dev_v_pheno + indiv));
				 
				 //check if retrieved geno is in high or low
				 index_in_table = index_to_int(geno, ORDER);
				 if (((*(&thread_table[0] + index_in_table )).cases)/((*(&thread_table[0] + index_in_table )).controls + 0.01) >= THR){
				 	if (ph)
						cases_high += 1;
					else
						controls_high += 1;
				 }
				 else{
				 	if (ph)
						cases_low += 1;
					else
						controls_low += 1;
				 }
				 	
				 
				 
				 /*
				 for (int i=0; i< TABLE_SIZE * ORDER; i++) {
				 	 if (high_genos[i] == 9){ //reached the end
				 	 	if (ph)
						 	cases_low += 1;
						 else
						 	controls_low += 1;
						 break;
						 }
				 	 	
				 	 int isequal = 1;
				 	 for (int j=0; j< ORDER; j++){
					 	if  (high_genos[i + j] != geno[i])
					 		isequal = 0;
					 		break;
					 	}
					 if (isequal){
					 	if (ph)
					 		cases_high += 1;
					 	else
					 		controls_high += 1;
					 	break; //found, exit loop;
					 	
					 }
			
					 	 
				 }
				 */
			}
	
		
		

			//test_measure = float(controls_high + cases_low)/float(cases_high + controls_high + cases_low + controls_low);
			if (MEASURE - '0' == 50){
			if (controls_low + cases_low == 0)
				//train_measure = float(controls_high + cases_low)/float(cases_high + controls_high + cases_low + controls_low);
				test_measure = (cases_high/float(cases_high + controls_high) + 0)/1;
			
			else if (cases_high + controls_high == 0)
				test_measure = (0 + controls_low/float(controls_low + cases_low))/1;
			else
				test_measure = (cases_high/float(cases_high + controls_high) + controls_low/float(controls_low + cases_low))/2;
		
			if (tid == TESTCOMB || (*(&thread_combination[0] + 0) == TESTSNP0 && *(&thread_combination[0] + 1) == TESTSNP1))
			printf("(tid %d) TRAIN BA %1.5f = (%d/(%d+%d) + %d/(%d+%d))/2\n", 
					 tid, test_measure, cases_high, cases_high, controls_high, controls_low, controls_low, cases_low);
			}
			else if (MEASURE - '0' == 49){
				test_measure = float(cases_high + controls_low)/float(cases_high + controls_low + controls_high + cases_low);
				if (tid == TESTCOMB || (*(&thread_combination[0] + 0) == TESTSNP0 && *(&thread_combination[0] + 1) == TESTSNP1))
				printf("(tid %d) TRAIN AC	 %1.5f = (%d+%d)/(%d+%d+%d+%d)\n", 
					 tid, test_measure, cases_high, controls_low, cases_high, controls_low, controls_high, cases_low);
			
			}
			//write result to global memory
			*(dev_output + NUMCOMBS * cv + 2 * tid + 1) = test_measure;
		
			 
			if (tid == TESTCOMB) printf("**********************************\n\n"); 
		
		}
	}
	
}

float* extract_min(struct str** top_cut_list, int cut, float* a){
	float minim = 1;
	float ind = -1;
	for (int i = 0; i < cut; i++){
		//fprintf(stderr,"inside func: value,index: %f : %d\n", (*(top_cut_list+i))-> value, (*(top_cut_list+i))-> index);
		if ((*(top_cut_list+i))-> value < minim){
			minim = (*(top_cut_list+i))-> value;
			ind = i;
		}
	}
	a[0] = minim;
	a[1] = ind;
	//fprintf(stderr,"%f : %f\n", *(a+0), *(a+1));
	return a;
}

void merge(struct str** arr, int l, int m, int r) 
{ 
    int i, j, k; 
    int n1 = m - l + 1; 
    int n2 =  r - m; 
  
    /* create temp arrays */
    struct str* L = (struct str*) malloc(n1 * sizeof(struct str));
    struct str* R = (struct str*) malloc(n2 * sizeof(struct str)); 
  
    /* Copy data to temp arrays L[] and R[] */
    for (i = 0; i < n1; i++) 
        *(L + i) = *(*(arr + l + i)); 
    for (j = 0; j < n2; j++) 
        *(R + j) = *(*(arr + m + 1+ j));
  
    /* Merge the temp arrays back into arr[l..r]*/
    i = 0; // Initial index of first subarray 
    j = 0; // Initial index of second subarray 
    k = l; // Initial index of merged subarray 
    while (i < n1 && j < n2) 
    { 
        if ((L+i)->value > (R + j)->value) 
        { 
            *(*(arr + k)) = *(L+i); 
            i++; 
        } 
        else
        { 
            *(*(arr + k)) = *(R+j); 
            j++; 
        } 
        k++; 
    } 
  
    /* Copy the remaining elements of L[], if there 
       are any */
    while (i < n1) 
    { 
        *(*(arr + k)) = *(L+i);
        i++; 
        k++; 
    } 
  
    /* Copy the remaining elements of R[], if there 
       are any */
    while (j < n2) 
    { 
        *(*(arr + k)) = *(R+j);  
        j++; 
        k++; 
    } 
    free(L);
    free(R);
} 
  
/* l is for left index and r is right index of the 
   sub-array of arr to be sorted */
void mergeSort(struct str** arr, int l, int r) 
{ 
    if (l < r) 
    { 
        // Same as (l+r)/2, but avoids overflow for 
        // large l and h 
        int m = l+(r-l)/2; 
  
        // Sort first and second halves 
        mergeSort(arr, l, m); 
        mergeSort(arr, m+1, r); 
  
        merge(arr, l, m, r); 
    } 
} 



void parseArgs(int argc, char **argv){
  int i=1;
  if(argc <= 3){
  
    printf("\nmissing input!\n");
	printf("\n\n");
    exit(0);
  }

  while(i<argc){
    if(!strcmp(argv[i], "-gf"))
      genoFile = argv[++i];
    else if(!strcmp(argv[i], "-cf"))
      combFile = argv[++i];
    else if(!strcmp(argv[i], "-pf"))
      phenoFile = argv[++i];
    else if(!strcmp(argv[i], "-out"))
      outputFile = argv[++i];
    else if(!strcmp(argv[i], "-basic_model"))
      basic_model = atoi(argv[++i]);
    else{
      printf("%s : argument not valid! \n",argv[i]);
      exit(1);
    }
    i++;
  }

  if( !genoFile || !phenoFile || !combFile || !outputFile){
    printf("no files specified	.. exiting\n");
    exit(1);
  }
  return;

}


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "ERROR HANDLED: %s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
  	}
  }
  
void readintData(char *dataFile, unsigned int rows, unsigned int cols, int * data){
  FILE *fp;
  int *dp = data;
  int i;

  fp = fopen(dataFile,"r");
  if(fp==NULL){
    fprintf(stderr,"error opening file.. exiting\n");
    //exit(1);
  } 
  
  for (i=0; i<rows*cols; ++i){
	  fscanf(fp, "%d", dp);
	  dp++;
  } 
  fclose(fp);
  return;
}


void readCombinations(char *dataFile, int rows, int cols, int * data){
  FILE *fp;
  int *dp = data;
  int i;

  fp = fopen(dataFile,"r");
  if(fp==NULL){
    fprintf(stderr,"error opening file.. exiting\n");
    //exit(1);
  } 
  
  for (i=0; i<rows*cols; ++i){
	  fscanf(fp, "%d", dp);
	  dp++;
  } 
  fclose(fp);
  return;
}
  	
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { printf( "Host memory failed in %s at line %d\n",  __FILE__, __LINE__ ); ( EXIT_FAILURE );}}

void print_cudaGetDeviceProperties(){
	hipDeviceProp_t prop;
	int count;
	HANDLE_ERROR( hipGetDeviceCount( &count ) );
	
	if (count == 0) {
        	fprintf(stderr,"error in print_cudaGetDeviceProperties: no devices supporting CUDA.\n");
        	return;
    	}
	
	for (int i=0; i< count; i++) {
		HANDLE_ERROR( hipGetDeviceProperties( &prop, i ) );
		printf( "   --- General Information for device %d ---\n", i );
		printf( "Name:  %s\n", prop.name );
		printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
		printf( "Clock rate:  %d\n", prop.clockRate );
		printf( "Device copy overlap:  " );
		if (prop.deviceOverlap)
		    printf( "Enabled\n" );
		else
		    printf( "Disabled\n");
		printf( "Kernel execution timeout :  " );
		if (prop.kernelExecTimeoutEnabled)
		    printf( "Enabled\n" );
		else
		    printf( "Disabled\n" );

		printf( "   --- Memory Information for device %d ---\n", i );
		printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
		printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
		printf( "Max mem pitch:  %ld\n", prop.memPitch );
		printf( "Texture Alignment:  %ld\n", prop.textureAlignment );

		printf( "   --- MP Information for device %d ---\n", i );
		printf( "Multiprocessor count:  %d\n",
		            prop.multiProcessorCount );
		printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
		printf( "Registers per mp:  %d\n", prop.regsPerBlock );
		printf( "Threads in warp:  %d\n", prop.warpSize );
		printf( "Max threads per block:  %d\n",
		            prop.maxThreadsPerBlock );
		printf( "Max thread dimensions:  (%d, %d, %d)\n",
		            prop.maxThreadsDim[0], prop.maxThreadsDim[1],
		            prop.maxThreadsDim[2] );
		printf( "Max grid dimensions:  (%d, %d, %d)\n",
		            prop.maxGridSize[0], prop.maxGridSize[1],
		            prop.maxGridSize[2] );
		printf( "\n" );
	}
	return;
}



void* routine( void *pvoidData) {
	DataStruct *data = (DataStruct*)pvoidData;
	HANDLE_ERROR( hipSetDevice( data->deviceID ) );
	
	int deviceID = data->deviceID;
	int deviceCount = data->deviceCount;
  	int* dev_mat_SNP;
  	int* dev_combinations;
	float* dev_output;
	int* dev_fp;
	int* dev_tp;
	
	int* mat_SNP = data->mat_SNP;
	int* combinations = data->combinations;
	int* v_pheno = data->v_pheno;
	int* cv_indices = data->cv_indices;
	float* output = data->output;
	int* tp = data->tp;
	int* fp = data->fp;
	clock_t start_clock = data->start_clock;
 	
  	//Allocate device memory
  	hipMalloc((void**)&dev_mat_SNP, mat_SNP_size);
  	
	//hipMalloc((void**)&dev_v_pheno, dev_v_pheno.mem_size); //no need, constant mem!
  	//hipMalloc((void**)&dev_cv_indices, indices_size); //no need, constant mem!
  	
  	// Copy host memory to device
  	//HANDLE_ERROR( hipMemcpy(dev_v_pheno, v_pheno, dev_v_pheno.mem_size, hipMemcpyHostToDevice));
  	hipMemcpyToSymbol(HIP_SYMBOL( dev_v_pheno),  v_pheno,  v_pheno_size );
  	hipMemcpyToSymbol(HIP_SYMBOL( dev_cv_indices),  cv_indices,  indices_size );
	HANDLE_ERROR( hipMemcpy(dev_mat_SNP, mat_SNP, mat_SNP_size, hipMemcpyHostToDevice));
	// (combinations_size / sizeof(int))/deviceCount) * deviceID is the same for all gpus, except last one is more loaded
	
	
	//start index (included), number of elements , end index (included)
	unsigned long s,d, e;
	s = (0 + (NUMCOMBS/deviceCount) * deviceID );
	if (deviceID + 1 < deviceCount)
		d = (NUMCOMBS/deviceCount);
	else
		d = NUMCOMBS - 1 - (s - 1); //how many total - how many done
	e = s + d - 1;
	hipMalloc((void**)&dev_combinations, (d *  ORDER * sizeof(int)) );
	HANDLE_ERROR( hipMemcpy(dev_combinations, (combinations + s * ORDER), (d *  ORDER * sizeof(int)), hipMemcpyHostToDevice));
	
	fprintf(stderr,"\nGPU %d, calling the kernel with this configuration:\n", deviceID);
	fprintf(stderr," comb-start: %lu, #combs: %lu, comb-end: %lu\n order: %d\n NSNPS: %d\n NIND: %d\n # CVs: %d\n THRESHOLD: %f\n BLOCK SIZE: %d\n GRID SIZE: %d\n",s,d,e, ORDER, NSNPS, NIND, CV, THR, BSx, GSx);

  	//HANDLE_ERROR( hipMemcpy(dev_cv_indices, cv_indices, indices_size, hipMemcpyHostToDevice));
  	//fprintf(stderr,"matrices copied  to GPU %d\n", deviceID);
  	
  	//hipHostAlloc((void**)&output,output.mem_size,hipHostMallocDefault);
  	
  	
  	s = (0 + ((NUMCOMBS *  ONEORTWO * CV)/(deviceCount) * deviceID ));
	if (deviceID + 1 < deviceCount)
		d = (NUMCOMBS *  ONEORTWO * CV)/(deviceCount);
	else
		d = (NUMCOMBS *  ONEORTWO * CV) - 1 - (s - 1); //how many total - how many done
	e = s + d - 1;
	//fprintf(stderr,"GPU %d, out-start: %lu, #out-values: %lu, out-end: %lu\n",deviceID,s,d,e);
	
	hipMalloc((void**)&dev_output, d * sizeof(float));
  	hipMalloc((void**)&dev_tp, d * sizeof(int));
  	hipMalloc((void**)&dev_fp, d * sizeof(int));
  	
	// kernel call
	dim3 dimBlock(BSx);//,BSy,BSz);
	dim3 dimGrid(GSx);//,GSy,GSz);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsedTime;
	hipEventRecord(start, 0);
	
	//fprintf(stderr,"GPU %d, calling kernel; elapsed %f seconds\n", deviceID, ((float)(clock() - start_clock) / CLOCKS_PER_SEC));
	
	MDR<<< dimGrid, dimBlock >>>(dev_mat_SNP, dev_output, dev_tp, dev_fp, dev_combinations, THR, deviceID, deviceCount);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	fprintf(stderr,"kernel computation terminated on GPU %d. GPU-Time required (ms): %4.5f\n", deviceID, elapsedTime);
	fprintf(stderr,"GPU %d, elapsed %f seconds\n", deviceID, ((float)(clock() - start_clock) / CLOCKS_PER_SEC));
	HANDLE_ERROR( hipEventDestroy( start ) );
	HANDLE_ERROR( hipEventDestroy( stop ) );
	
	
	hipMemcpy(output + s, dev_output, d * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(tp + s, dev_tp, d * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(fp + s, dev_fp, d * sizeof(int), hipMemcpyDeviceToHost);
	
	//fprintf(stderr,"GPU%d, output, tp, fp copied to host \n", deviceID);
	
	
  	//free
  	hipFree(dev_mat_SNP);
	//hipFree(dev_v_pheno);
	hipFree(dev_output);
	hipFree(dev_tp);
	hipFree(dev_fp);
	hipFree(dev_combinations);
	//hipFree(dev_cv_indices);
	//printf("GPU %d, returning; \nelapsed %f seconds\n", deviceID, ((float)(clock() - start_clock) / CLOCKS_PER_SEC));
	return 0;
}

/************************/
//MAIN
/************************/

int main(int argc, char **argv)
{

	clock_t start_clock = clock();
	
	
	//Parsing the input parameters
	parseArgs(argc,argv);
	 
	 
  	//print_cudaGetDeviceProperties(); 
  	
  	int deviceCount;
	HANDLE_ERROR( hipGetDeviceCount( &deviceCount ) );
	//printf( "found %d devices\n", deviceCount );
	if (NUMDEVICES > deviceCount || deviceCount == 0){
		fprintf(stderr,"error: less devices detected (%d) than specified (%d)! Exiting...\n", deviceCount, NUMDEVICES);
		return 0;
	}
	//fprintf(stderr,"Gonna use first %d of %d devices\n", NUMDEVICES, deviceCount);
	deviceCount = imin(deviceCount, NUMDEVICES);
  	
  	hipDeviceProp_t prop;
  	HANDLE_ERROR( hipGetDeviceProperties( &prop, 0 ) );
  	
  	if (NUMCOMBS/deviceCount > prop.maxGridSize[0])
  		fprintf(stderr,"Supported up to %d*%d combs. input'll be considered up to that combination. Run again with new file later.\n", deviceCount, prop.maxGridSize[0]);
  	
  	
  	//fprintf(stderr,"\n*****************\n");
	//fprintf(stderr,"Multifactor Dimensionality Reduction\n");
	//fprintf(stderr,"*****************\n\n");	
	
	//Allocate host memory 
	int* mat_SNP = (int*)malloc(mat_SNP_size); 
	int* v_pheno = (int*)malloc(v_pheno_size);
	float* output = (float*)malloc(output_size);
	int* tp = (int*)malloc(tp_size);
	int* fp = (int*)malloc(fp_size);
	int* combinations = (int*)malloc(combinations_size);
	int* cv_indices = (int*)malloc(indices_size);
	
	/*
	if (CV <= 1){
		fprintf(stderr,"will run only one pass, no train-test... \n");
	}
	*/
	//generate a permutation of the individuals indices
	for(int i=0;i<NIND;++i){
        	*(cv_indices + i) = i;
    		}
    	
    	/*	
		//permute r with Fisher-Yates shuffling algorithm
	for (int i = NIND; i >= 0; --i){
		//generate a random number [0, n-1]
		int j = rand() % (i+1);

		//swap the last element with element at random index
		int temp = *(cv_indices + i);
		*(cv_indices + i) = *(cv_indices + j);
		*(cv_indices + j) = temp;
	}
	*/
  	
  	//Read the matrix in host data
	readintData(genoFile, NSNPS, NIND, mat_SNP);
	//fprintf(stderr,"geno file read..\n");
	readintData(phenoFile, NIND, 1, v_pheno);
	//fprintf(stderr,"pheno file read..\n");
	
	int ncases;
	if (THR < 0){
		ncases = 0;
		for (int i = 0; i < NIND; i++){
			if ( *(v_pheno + i) )
				ncases += 1;
		}
		THR = float(ncases)/(NIND - ncases);
		printf("no input threshold; automatically set to %f = %d/%d \n", THR, ncases, NIND-ncases);
	}
	
	
	if (basic_model % 2 != 0){
		
		//fprintf(stderr,"detected NULL model. Exhaustive in-memory generation...\n");
		int t = 0;
		if (ORDER == 2) {
			for(int i=0;i<NSNPS;++i){
				for(int j=i+1;j<NSNPS;++j){
					combinations[t] = i;
					combinations[t+1] = j;
					t += ORDER;
					
				}
			}
		}
		else if (ORDER == 3) {
			for(int i=0;i<NSNPS;++i){
				for(int j=i+1;j<NSNPS;++j){
					for(int k=j+1;k<NSNPS;++k){
							combinations[t] = i;
							combinations[t+1] = j;
							combinations[t+2] = k;
							t += ORDER;
					}
				}
			}
		
		}
		else if (ORDER == 4) {
			for(int i=0;i<NSNPS;++i){
				for(int j=i+1;j<NSNPS;++j){
					for(int k=j+1;k<NSNPS;++k){
						for(int l=k+1;l<NSNPS;++l){
							combinations[t] = i;
							combinations[t+1] = j;
							combinations[t+2] = k;
							combinations[t+3] = l;
							t += ORDER;
						}
					}
				}
			}
		}
		else if (ORDER == 5) {
			for(int i=0;i<NSNPS;++i){
				for(int j=i+1;j<NSNPS;++j){
					for(int k=j+1;k<NSNPS;++k){
						for(int l=k+1;l<NSNPS;++l){
							for(int m=l+1;m<NSNPS;++m){
								combinations[t] = i;
								combinations[t+1] = j;
								combinations[t+2] = k;
								combinations[t+3] = l;
								combinations[t+4] = m;
								t += ORDER;
							}
						}
					}
				}
			}
		}
		else if (ORDER == 6) {
			for(int i=0;i<NSNPS;++i){
				for(int j=i+1;j<NSNPS;++j){
					for(int k=j+1;k<NSNPS;++k){
						for(int l=k+1;l<NSNPS;++l){
							for(int m=l+1;m<NSNPS;++m){
								for(int n=m+1;n<NSNPS;++n){
									combinations[t] = i;
									combinations[t+1] = j;
									combinations[t+2] = k;
									combinations[t+3] = l;
									combinations[t+4] = m;
									combinations[t+5] = m;
									t += ORDER;
								}
							}
						}
					}
				}
			}
		}
	//fprintf(stderr,"combinations in-memory generated..\n");	
	}
	else{	
		
		//Read combinations
		fprintf(stderr,"detected disease model\n");
		readCombinations(combFile, NUMCOMBS, ORDER, combinations);
		fprintf(stderr,"combinations file read..\n");
	}
	

			
  	//printf("calling therads; \nelapsed %f seconds\n", ((float)(clock() - start_clock) / CLOCKS_PER_SEC));
  	CUTThread threads[deviceCount-1];
  	DataStruct  data[deviceCount];
  	
  	for(int i=0;i<deviceCount;++i){
  		data[i].deviceID = i;
  		data[i].deviceCount = deviceCount;
		data[i].mat_SNP = mat_SNP;
		data[i].combinations = combinations; //same for all. split happens in the CUDAMemCpy
		data[i].v_pheno = v_pheno;
		data[i].cv_indices = cv_indices;
		data[i].output = output;
		data[i].tp = tp;
		data[i].fp = fp;
		data[i].start_clock = start_clock;
  		if (i != (deviceCount -1)){
  			threads[i] = start_thread( routine, &(data[i]) );
  			
  		}
  		else{
  			routine( &(data[i]) );
  			for(int j=0;j<deviceCount-1;++j)
				end_thread( threads[j] );
		}
  	}
  	
	//printf("last thread terminated. \nelapsed %f seconds\n", ((float)(clock() - start_clock) / CLOCKS_PER_SEC));
	free(mat_SNP);
	free(v_pheno);
	free(cv_indices);
	
	
	if (strcmp(outputFile, "no_out")){
		//sort output and print to file
		struct str **array = (struct str **) malloc(NUMCOMBS * sizeof(struct str*));
		
	
		FILE *fpout;
		fpout = fopen(outputFile, "w");
	
		if (CV == 1){
			//fprintf(fpout,"---------- measure ----------\n");
			
			for(int i=0;i<NUMCOMBS;i++){
				*(array + i) = (struct str*) malloc(sizeof(struct str));
				(*(array + i))->value = *(output + i);
				(*(array + i))->index = i;
				//objects[i].value=*(output + i);
				//objects[i].index=i;
			}
			//printf("last thread terminated. \nelapsed %f seconds\n", ((float)(clock() - start_clock) / CLOCKS_PER_SEC));
			fprintf(stderr,"Effectively sorting output..elapsed %f seconds\n", ((float)(clock() - start_clock) / CLOCKS_PER_SEC));
			

			//fprintf(stderr,"\nelapsed %f seconds\n", ((float)(clock() - start_clock) / CLOCKS_PER_SEC));
		       	//fprintf(stderr,"Effectively writing to file: %s \n", outputFile);
		       	int cut = imin(CUT,NUMCOMBS);

			/*
			//version merge all

		       	mergeSort(array, 0, NUMCOMBS - 1);	
		       	fprintf(stderr,"terminated merge sort.. elapsed %f seconds\n", ((float)(clock() - start_clock) / CLOCKS_PER_SEC));
		       	//fprintf(stderr,"breaking to first %d combinations", cut);
			for (int j = 0; j < cut; j++){
				for (int q=0; q< ORDER; q++){
					if (q == 0)
						fprintf(fpout,"snp%d ", *(combinations + (*(array + j))->index * ORDER + q));
					else if (q == ORDER -1){
					
						int tp_val = *(tp + (*(array + j))->index);
						int fp_val = *(fp + (*(array + j))->index);
						int tn_val = NIND - ncases - fp_val;
						int fn_val = ncases - tp_val;
						
						fprintf(fpout,"snp%d %f %d %d %d %d\n", 
							//*(combinations + j * ORDER + q), *(output + NUMCOMBS * (CV -1) + 1 * objects[j].index + 0));
							//*(combinations + j * ORDER + q), objects[j].value);
							*(combinations +(*(array + j))->index * ORDER + q), (*(array + j))->value, tp_val, fp_val, tn_val, fn_val);
					}
					else
						fprintf(fpout,"snp%d ", *(combinations + (*(array + j))->index * ORDER + q));
					
			
				}
			}
			//end version merge all
			
			*/
			
			
			
			//alternative
			//new version
		       	struct str **top_cut = (struct str **) malloc(cut * sizeof(struct str*));
		       	for (int j = 0; j < cut; j++){
		       		*(top_cut + j) = (struct str*) malloc(sizeof(struct str));
		       		(*(top_cut + j))->value = 0;
		       	}
		       	
		       	//scan through output and insert in top_cut
		       	float min_cut = 0;
		       	int ind;
		       	float* a = (float*) malloc(2 * sizeof(float));
		       	
		       	for (int j = 0; j < NUMCOMBS; j++){
		       		
		       		
		       		if (*(output + j) < min_cut)
		       			continue;
		       		//fprintf(stderr,"---\n");
		       		//fprintf(stderr,"min_cut,index: %f %d\n", min_cut, ind);
		       		else {
		       			
		       			extract_min(top_cut, cut, a);
			       		//fprintf(stderr,"%f, %f", *(a+0), *(a+1));
			       		min_cut =*(a+0);
			       		ind = (int) *(a+1);
		       			
		       			//fprintf(stderr,"removing %f %d\n,", min_cut, ind);
		       			(*(top_cut + ind))-> index = j;
		       			(*(top_cut + ind))-> value = *(output + j);
		       			//fprintf(stderr,"inserting %f %d",*(output + j), j);
		       			//for (int r = 0; r < cut; r++){
		       				//fprintf(stderr,"index r-th,value r-th: %d,%f \n", (*(top_cut + r))-> index, (*(top_cut + r))-> value);
		       			//fprintf(stderr,"\n");
		       			//}
		       			//fprintf(stderr,"inserted output + j at index j: %f, %d\n", *(output + j), j);
		       		}
		       	}

		       	fprintf(stderr,"top cut sorting... \n");
		       	
		       mergeSort(top_cut, 0, cut - 1);	
			
			for (int j = 0; j < cut; j++)
		       		fprintf(stderr,"index,value: %d,%f \n", (*(top_cut + j))-> index, (*(top_cut + j))-> value);
		       fprintf(stderr,"top cut sorted; elapsed %f seconds\n", ((float)(clock() - start_clock) / CLOCKS_PER_SEC));
		       
		       
		       
		       for (int j = 0; j < cut; j++){
				for (int q=0; q< ORDER; q++){
					if (q == 0)
						fprintf(fpout,"snp%d ", *(combinations + (*(top_cut + j))->index * ORDER + q));
					else if (q == ORDER -1){
					
						int tp_val = *(tp + (*(top_cut + j))->index);
						int fp_val = *(fp + (*(top_cut + j))->index);
						int tn_val = NIND - ncases - fp_val;
						int fn_val = ncases - tp_val;
						
						fprintf(fpout,"snp%d %f %d %d %d %d\n", 
							//*(combinations + j * ORDER + q), *(output + NUMCOMBS * (CV -1) + 1 * objects[j].index + 0));
							//*(combinations + j * ORDER + q), objects[j].value);
							*(combinations +(*(top_cut + j))->index * ORDER + q), (*(top_cut + j))->value, tp_val, fp_val, tn_val, fn_val);
					}
					else
						fprintf(fpout,"snp%d ", *(combinations + (*(top_cut + j))->index * ORDER + q));
					
			
				}
			}
			
			//end new version
			
			
		
		/*
		else{
	  		for (int cv = 0; cv < CV; cv++){
		  		fprintf(fpout,"---------- CV %d/%d train_measure test_measure(s) ----------\n", cv+1, CV);
		  		
		  		for(int i=0;i<NUMCOMBS;i++){
					objects[i].value=*(output + NUMCOMBS * cv + 2 * i + 1); //sorting on test measure!
					objects[i].index=i;
				}
				qsort(objects,NUMCOMBS,sizeof(objects[0]),cmp);

		  		for (int j = 0; j < NUMCOMBS; j++){
					for (int q=0; q< ORDER; q++){
						if (q == 0)
							fprintf(fpout,"snp%d ", *(combinations + j * ORDER + q));
						else if (q == ORDER -1)
							fprintf(fpout,"snp%d %f %f\n", 
								*(combinations + j * ORDER + q),
								*(output +  NUMCOMBS * cv + 2 * objects[j].index + 0), //sorted on test measure!
								*(output + NUMCOMBS * cv + 2 * objects[j].index + 1) );
								//*(output +  NUMCOMBS * cv + 2 * j + 0),
								//*(output + NUMCOMBS * cv + 2 * j + 1) );
						else
							fprintf(fpout,"snp%d ", *(combinations + j * ORDER + q));
					}

				}
			}
		}*/
		//fprintf(stderr,"Output written to file %s\n", outputFile);
		}
	}
	
	//else
		//fprintf(stderr,"Output was not saved to file \n");
	
	free(output);
	free(tp);
	free(fp);
   	free(combinations);

 	return 0;
}
